#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"
#include <iostream>
#include <vector>
#include <array>
#include <algorithm>
#include "spmv_seg.cu"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////
inline int remove_excluded_pages_from_matrix(bool *excluded, std::vector<int> &x, std::vector<int> &y, std::vector<double> &val, int E);

// Write GPU kernel here!

__inline__ __device__ void warp_reduction(volatile double *input, int threadId)
// we need volatile flag here, otherwise the compiler might introduce some optimizations in the "input" variable
// and place it in registers instead of shared memory!
{
	input[threadId] += input[threadId + 32];
	input[threadId] += input[threadId + 16];
	input[threadId] += input[threadId + 8];
	input[threadId] += input[threadId + 4];
	input[threadId] += input[threadId + 2];
	input[threadId] += input[threadId + 1];
}

__device__ void collect_res_gpu(double *input, int numOfBlocks) // compute the final reduction
{
    int i, threadId = threadIdx.x;

    /*
    for (i = 0; i < numOfBlocks; i += blockDim.x) // collect the result of the various blocks
    {
        if ((threadId + i) * blockDim.x < numOfBlocks){
            localVars[threadId] += input[(threadId + i) * blockDim.x];
        }
        __syncthreads();
    }
    */

    for (i = blockDim.x / 2; i > 32; i >>= 1) // compute the parallel reduction for the collected data
    {
        if (threadId < i)
        {
            input[threadId] += input[threadId + i];
        }
        __syncthreads();
    }

    if(threadId<32)
        warp_reduction(input,threadId);
    __syncthreads();

    /*
    if(threadId==0)
        input[threadId] = localVars[threadId];
    __syncthreads();
    */
    
}

/**
 * @brief Parallel GPU version of matrix-vector multiplication.
 * 
 * @param x row indices (COO format matrix - vector)
 * @param y column indices (COO format matrix - vector)
 * @param val matrix values (COO format matrix - vector)
 * @param vec vector
 * @param result vector for result of the multiplication
 * @param N vector dimension
 */
__global__ void spmv_coo_0(const int *x, const int *y, const double *val, const double *vec, double *result, int N) {
    // Uses a grid-stride loop to perform dot product
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        atomicAdd(result + x[i], val[i] * vec[y[i]]);
    }
}

__device__ int binary_search_element(const int elem, const int *arr, int low, int high){
    int mid;
    while( low != high ){
        mid = (low + high)/2;
        if (elem == arr[mid])
        return mid;

        else if (elem > arr[mid]) // elem is on the right side
            low = mid + 1;

        else                  // elem is on the left side
            high = mid - 1;
    }
    return -1; //not found
}

__global__ void spmv_coo_1(const int *x, const int *y, const double *val, const double *vec, double *result, const int N, const int res_size, int *shrinked_x){
    extern __shared__ int s[]; //must be initialized with N elements from the caller

    int *temp_idx = s;
    double *temp_res = (double*)&temp_idx[res_size];
    //int *last_idx = (int*)&temp_res[res_size];
    int binary_search_res;


    for (int i = threadIdx.x, j = blockIdx.x * blockDim.x + threadIdx.x; i < res_size; i += blockDim.x, j += blockDim.x * gridDim.x){ 
        temp_res[i] = 0.0;
        temp_idx[i] =shrinked_x[j];
    }

    __syncthreads();

    // Uses a grid-stride loop to perform dot product
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        binary_search_res = binary_search_element(x[i], temp_idx, 0, res_size-1);
        atomicAdd_block(temp_res + binary_search_res, (val[i] * vec[y[i]])); //only thread-safe, not block-safe
    }

    __syncthreads();
    //riassamble all the partial results: the speedup should be that we're doing num_blocks*V atomicAdd() instead of num_blocks*E atomicAdd()
    for(int i=threadIdx.x; i < res_size; i += blockDim.x){
        atomicAdd(&result[temp_idx[i]], temp_res[i]);
    }

}

//highly dependent on how the kernel accesses vector x
void compute_shrinked_x(std::vector<int> &shrinked_x, const int threadsPerBlock, const int blocksPerGrid, std::vector<int> x, const int E){
    int threadIdx = 0;
    int blockIdx = 0;
    int last_idx = -1;
    int last_block = 0;

    if(threadsPerBlock < 1 || blocksPerGrid < 1){
        return;
    }

    for(int i=0; i < E; i++){
        if(!(last_idx == x[i] && blockIdx == last_block)){
            shrinked_x.push_back(x[i]);
        }

        threadIdx = (threadIdx + 1) % threadsPerBlock;
        if(threadIdx == 0){
            last_block = blockIdx;
            blockIdx = (blockIdx + 1) % blocksPerGrid;
        }
    }

    //warning: if a single page is present in x enough times so that it is repeated even in the second call of a same block, then we will have duplicates in temp_idx in spmv_coo_1

}

/**
 * @brief Heuristi version. Compute spmv just for the first precision% (e.g. 80%) vertex
 * 
 * @return __global__ 
 */
__global__ void spmv_coo_h(const int *x, const int *y, const double *val, const double *vec, double *result, int N, const bool *excluded_pages){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        if(!excluded_pages[x[i]])
            atomicAdd(result + x[i], val[i] * vec[y[i]]);
    }
}


__global__ void lower_pr_unselected_pages_h(double *result, bool *excluded_pages, int N){
    for(int i= blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        if(excluded_pages[i]){
            result[i] = 0;
        }
}

//////////////////////////////
//////////////////////////////

// CPU Utility functions;

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            printf("Err: %s\n", hipGetErrorName(err));                                   \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
        val[i] = 1.0 / outdegree[y[i]];  
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph(); //CPU loading data

    // Allocate any GPU data here;
    // TODO!

    // Allocate GPU data: cloning x, y and val vectors in GPU global memory
    CHECK(hipMalloc(&x_d, sizeof(int)*E));
    CHECK(hipMalloc(&y_d, sizeof(int)*E));
    CHECK(hipMalloc(&val_d, sizeof(double)*E));
    CHECK(hipMalloc(&pr_gpu, sizeof(double)*V));  
    CHECK(hipMalloc(&gpu_result, sizeof(double)*V));
    CHECK(hipMalloc(&gpu_err, sizeof(double)));
    CHECK(hipMalloc(&dangling_factor_gpu, sizeof(double)));
    CHECK(hipMalloc(&dangling_bitmap, sizeof(int)*dangling.size()));
    CHECK(hipMalloc(&excluded_pages_gpu, sizeof(bool) * V));

    CHECK(hipMemcpy(x_d, &x[0], sizeof(int) * x.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(y_d, &y[0], sizeof(int) * y.size(), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(val_d, &val[0], sizeof(double) * val.size(), hipMemcpyHostToDevice));
    
}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;
    // TODO!

    int last_idx = -1;
    int last_idx_qty = 0;


    std::vector<std::pair<int, int>> in_degree;
    in_degree.resize(V);

    //assumption: E > V
    for(int i=0; i<E; i++){
        //compute in_degree for each page: if page i not it x[i] --> in_degree[i] = 0
        if(x[i] != last_idx){
            if(last_idx >= 0){
                in_degree[last_idx].first = last_idx_qty;
                in_degree[last_idx].second = last_idx;
            }

            last_idx++;
            last_idx_qty = (x[i] != last_idx) ? 0 : 1;

        }else{
            last_idx_qty++;
        }
    }
    in_degree[last_idx].first = last_idx_qty; // finish writing the qty for the last page
    in_degree[last_idx].second = last_idx;

    while(last_idx < V){
        // last pages are not in x, so finish up in_degree with zeros
        last_idx++;
        in_degree[last_idx].first = 0;
        in_degree[last_idx].second = last_idx;
    }

    auto greater_key = [](std::pair<int,int> e1, std::pair<int,int> e2){
        return e1.first > e2.first;
        };

    // sort in_degree_ranked
    // key: in_degree   values: in_degree_ranked    comparer: reverting
    sort(in_degree.begin(), in_degree.end(), greater_key);


    int precise_vertex_qty = (int)(V * heuristic_precision);
    std::cout << "Precision: " << heuristic_precision << ", #precise vertex: " << precise_vertex_qty << std::endl;

    //excluded_pages_cpu.resize(V);
    excluded_pages_cpu = (bool*)malloc(sizeof(bool) * V);
  
    //std::cout << "Excluded pages:" << std::endl;
    for(int i=0; i<V; i++){
        excluded_pages_cpu[in_degree[i].second] = (i< precise_vertex_qty && in_degree[i].first != 0) ? false : true;
    }

    num_effective_vertex = std::count(excluded_pages_cpu, &excluded_pages_cpu[V-1], false);

    CHECK(hipMemcpy(excluded_pages_gpu, excluded_pages_cpu, sizeof(bool) * V, hipMemcpyHostToDevice));
}

/**
 * @brief It modify the adjecency matrix (in COO format) by removing the edges that represent a link from page i to page j where i has to be excluded.
 * 
 * @param excluded array that specify for each page (dim = V) is it has to be excluded (true) or not (false)
 * @param x row vector of matrix
 * @param y column vector of matrix
 * @param val value vector of matrix
 * @param E current number of edges in the graph/matrix
 * @return updated integer number of edges in the graph after the removal
 */
inline int remove_excluded_pages_from_matrix(bool *excluded, std::vector<int> &x, std::vector<int> &y, std::vector<double> &val, int E){
    int num_matrix_edges = E;

    for(int i=0; i < num_matrix_edges; i++){
        if(excluded[y[i]]){
            // page x[i] is linked by a page with in_degree = 0 --> we remove the "link" from the matrix
            x.erase(x.begin() + i);
            y.erase(y.begin() + i);
            val.erase(val.begin() + i);
            num_matrix_edges--;
            i--;
        }
    }

    return num_matrix_edges;
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    //std::fill(pr.begin(), pr.end(), 1.0 / V); 
    // Generate a new personalization vertex for this iteration;
    
    
    double penalty_weight = 0.1;
    float weighted_sum = num_effective_vertex * (1 - penalty_weight) + (V - num_effective_vertex) * penalty_weight;

    for(int i=0; i<V; i++){
        if(excluded_pages_cpu[i]){
            pr[i] = penalty_weight * (1/weighted_sum);
        }else{
            pr[i] = (1-penalty_weight) * (1/weighted_sum);
        }
    }
    
    

    personalization_vertex = rand() % V; 
    if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    // Do any GPU reset here, and also transfer data to the GPU;
    // TODO!

    CHECK(hipMemcpy(pr_gpu, &pr[0], sizeof(double) * pr.size(), hipMemcpyHostToDevice));
    // initialize GPU result array with all 0s
    CHECK(hipMemset(gpu_result, 0.0, sizeof(double)*V));
    CHECK(hipMemcpy(dangling_bitmap, dangling.data(), sizeof(int)*dangling.size(), hipMemcpyHostToDevice));


    printf("Is personalization vertex %d in discarded pages? %s\n", personalization_vertex, excluded_pages_cpu[personalization_vertex] ? "true" : "false");
}


/**
 * @brief Computation for the dangling factor.
 * 
 * @param a dangling seed vector
 * @param b page ranking vector
 * @param N vector size
 * @param result pointer to the dangling factor (where to place the result of the function)
 */
__global__ void compute_dangling_factor_gpu(const int *a, const double *b, const int N, double *result){
    // using a share temp_result might speed up but we have problems in sync. the blocks
    extern __shared__ double temp[];
    temp[threadIdx.x]=0;
    __syncthreads();

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        temp[threadIdx.x] = a[idx] * b[idx];
        //atomicAdd(temp + threadIdx.x, a[idx] * b[idx]);
    }
    __syncthreads();

    collect_res_gpu(temp, blockDim.x);
    if (threadIdx.x == 0) {
        atomicAdd(result, temp[0]);
    }
}

/**
 * @brief Final formula for PR
 * 
 * @param alpha damping factor
 * @param x vector of intermediate pr values
 * @param beta damping factor: alpha * damping_factor / V
 * @param result final vector of the PR value results
 * @param N vectors dimension
*/
__global__ void axpb_personalized_gpu(double alpha, double *x, double beta, const int personalization_vertex, double *result, const int N){
    __shared__ double one_minus_alpha;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIdx.x == 0)
        one_minus_alpha = 1 - alpha;
    
    __syncthreads();

    for(; i < N; i+= blockDim.x * gridDim.x){
        result[i] = alpha * x[i] + beta + ((personalization_vertex == i) ? one_minus_alpha : 0.0);
    }
}

/**
 * @brief GPU parallelized version for euclidean distance
 * 
 * @param x vector of x (row) coordinates
 * @param y vector of y (column) coordinates
 * @param N dimension of the vectors
 * @param result pointer to the computed distance result
 */
__global__ void euclidean_distance_gpu(const double* x , const double* y , const int N, double* result, bool *excluded_pages) {
    extern __shared__ double temp[];
    temp[threadIdx.x]=0;
    __syncthreads();
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double var;

    if (idx < N) {
        var = x[idx] - y[idx];
        //TODO: which one is faster? do i need atomic op. with reduction?
        //atomicAdd(temp + threadIdx.x, var * var);
        //temp[threadIdx.x] = __pow(var, 2);
        temp[threadIdx.x] = var * var;
    }
    __syncthreads();
    
    collect_res_gpu(temp, blockDim.x);
    if (threadIdx.x == 0) {
        atomicAdd(result, temp[0]);
    }
}

void PersonalizedPageRank::personalized_pagerank_0(int iter){
    auto start_tmp = clock_type::now();
 
    int blockSize = block_size; // take block size from option -t
    int gridSize = (E + blockSize - 1) / blockSize;

    int blockSize_shared = 256;
    int gridSize_shared = (V + blockSize_shared -1)/ blockSize_shared;

    std::cout << "blockSize: " << blockSize << "\tgridSize: " << gridSize << std::endl;
    std::cout << "blockSize_shared: " << blockSize_shared << "\tgridSize_shared: " << gridSize_shared << std::endl;

    dim3 blocksPerGrid(gridSize, 1, 1);
    dim3 threadsPerBlock(blockSize, 1, 1);

    dim3 blocksPerGridShared(gridSize_shared, 1, 1);
    dim3 threadsPerBlockShared(blockSize_shared, 1, 1);

    double *temp, dangling_factor_val;
    double *err = (double *) malloc(sizeof(double));         // convergence error

    //printf("blocksPerGrid: %d\tthreadsPerBlock:%d\n", (E + blocksize - 1) / blocksize, blocksize);
    hipStream_t spmv_stream, dangling_factor_stream, lower_pr_stream;
    hipStreamCreate(&spmv_stream);
    hipStreamCreate(&dangling_factor_stream);
    hipStreamCreate(&lower_pr_stream);

    

    int number_of_iterations = 0;
    bool conv = false;
    while (!conv && number_of_iterations < max_iterations) {
        //std::cout << "Inner iteration #" << number_of_iterations << std::endl;
        CHECK(hipMemset(gpu_result, 0.0, sizeof(double) * V));    // reset GPU result

        //CHECK(hipMemsetAsync(dangling_factor_gpu, 0.0, sizeof(double), dangling_factor_stream));  
        CHECK(hipMemset(gpu_err, 0.0, sizeof(double)));             // reset error 
        //hipMemset(dangling_factor_gpu, 0.0, sizeof(double));      // reset dangling factor

        //spmv_coo_0<<<blocksPerGrid, threadsPerBlock, 0 , spmv_stream>>>(x_d, y_d, val_d, pr_gpu, gpu_result, E);
        __spmv_coo_flat(x_d, y_d, val_d, pr_gpu, gpu_result, E, spmv_stream);
        //CHECK(hipDeviceSynchronize());
        //hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        //spmv_coo_1<<<blocksPerGrid, threadsPerBlock, V*sizeof(float)>>>(x_d, y_d, val_d, pr_gpu, gpu_result, E, V);
        //spmv_coo_h<<<blocksPerGrid, threadsPerBlock, 0, spmv_stream>>>(x_d, y_d, val_d, pr_gpu, gpu_result, E, excluded_pages_gpu);
        CHECK_KERNELCALL();

        //lower_pr_unselected_pages_h<<<blocksPerGrid, threadsPerBlock, 0, lower_pr_stream>>>(gpu_result, excluded_pages_gpu, V);
        //CHECK_KERNELCALL();

        compute_dangling_factor_gpu<<<blocksPerGrid, threadsPerBlock, blockSize * sizeof(double), dangling_factor_stream>>>(dangling_bitmap, pr_gpu, V, dangling_factor_gpu);
        //compute_dangling_factor_h<<<blocksPerGrid, threadsPerBlock, blockSize * sizeof(double), dangling_factor_stream>>>(dangling_bitmap, pr_gpu, V, dangling_factor_gpu, excluded_pages_gpu);
        CHECK_KERNELCALL();

        //on wiki si rompe la dangling ( o forse ancora prima?)

        CHECK(hipMemcpy(&dangling_factor_val, dangling_factor_gpu, sizeof(double), hipMemcpyDeviceToHost));

        axpb_personalized_gpu<<<blocksPerGrid, threadsPerBlock>>>(custom_alpha, gpu_result, custom_alpha * dangling_factor_val / V, personalization_vertex, gpu_result, V);
        //axpb_personalized_h<<<blocksPerGrid, threadsPerBlock>>>(custom_alpha, gpu_result, custom_alpha * dangling_factor_val / V, personalization_vertex, gpu_result, V, excluded_pages_gpu);
        CHECK_KERNELCALL();

        
        CHECK(hipMemsetAsync(dangling_factor_gpu, 0.0, sizeof(double), dangling_factor_stream));  // asynchronously reset the dangling factor on the GPU
        
        // Check convergence
        euclidean_distance_gpu<<<blocksPerGrid, threadsPerBlock, blockSize * sizeof(double)>>>(pr_gpu, gpu_result, V, gpu_err, excluded_pages_gpu);
        //euclidean_distance_h<<<blocksPerGrid, threadsPerBlock, blockSize * sizeof(double)>>>(pr_gpu, gpu_result, V, gpu_err, excluded_pages_gpu);
        CHECK_KERNELCALL();


        hipMemcpy(err, gpu_err, sizeof(double), hipMemcpyDeviceToHost);
        *err = std::sqrt((double) *err);
        conv = *err <= convergence_threshold;

        temp = pr_gpu;
        pr_gpu = gpu_result;
        gpu_result = temp; 

        //printf("ITER: %d - ERR: %lf\n", number_of_iterations, *err);

        number_of_iterations++;
    }
    std::cout << "Total inner iteration #" << number_of_iterations << std::endl;

    CHECK(hipDeviceSynchronize());

    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (3 * sizeof(double) * N * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // destroy the streams
    hipStreamDestroy(spmv_stream);
    hipStreamDestroy(dangling_factor_stream);
    hipStreamDestroy(lower_pr_stream);

    // save the GPU PPR values into the "pr" array
    CHECK(hipMemcpy(&pr[0], pr_gpu, sizeof(double) * V, hipMemcpyDeviceToHost));
}

void PersonalizedPageRank::execute(int iter) {
    // Do the GPU computation here, and also transfer results to the CPU;
    switch (implementation)
    {
    case 0:
        personalized_pagerank_0(iter);
        break;
    default:
        break;
    }
}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    // TODO!
    
    hipFree(x_d);
    hipFree(y_d);
    hipFree(val_d);
    hipFree(pr_gpu);
    hipFree(gpu_result);
    hipFree(gpu_err);
    hipFree(dangling_factor_gpu);
    hipFree(dangling_bitmap);
    hipFree(excluded_pages_gpu);


    free(excluded_pages_cpu);
}
